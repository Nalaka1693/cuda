
#include <hip/hip_runtime.h>
#include <stdio.h>

#define SIZE 8

__global__ void addVector(int vectorAns[SIZE], int vectorA[SIZE], int vectorB[SIZE]);

int main() {

    int vectorA[SIZE];
    int vectorB[SIZE];
    int vectorAns[SIZE];

    int i;
    for (i = 0; i < SIZE; i++) {
        vectorA[i] = i;
        vectorB[i] = SIZE - i;
    }

    int *d_A;
    int *d_B;
    int *d_C;

    //allocate memmory
    hipMalloc((void **) &d_A, sizeof(int) * SIZE);
    hipMalloc((void **) &d_B, sizeof(int) * SIZE);
    hipMalloc((void **) &d_C, sizeof(int) * SIZE);

    //copy inputs from RAM to GPU
    hipMemcpy(d_A, vectorA, sizeof(int) * SIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_B, vectorB, sizeof(int) * SIZE, hipMemcpyHostToDevice);

    //calculation function
    addVector<<< 1, SIZE >>> (d_C, d_A, d_B);

    //copy back to RAM
    hipMemcpy(vectorAns, d_C, sizeof(int) * SIZE, hipMemcpyDeviceToHost);

    //cuda free
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);


    printf("Answer is : \n");

    for (i = 0; i < SIZE; i++) {
        printf("%d ", vectorAns[i]);
    }
    printf("\n");

    return 0;
}

__global__ void addVector(int vectorAns[SIZE], int vectorA[SIZE], int vectorB[SIZE]) {
    int i = threadIdx.x;
    vectorAns[i] = vectorA[i] + vectorB[i];

}
