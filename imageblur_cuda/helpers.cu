/* C file that implements the helper functions specified in helpers.cu */

#include <stdio.h>
#include "helpers.cuh"
	
/* check whether the last CUDA function or CUDA kernel launch is erroneous and if yes an error message will be printed
and then the program will be aborted*/

void gpuAssert(const char *file, int line){

	hipError_t code = hipGetLastError();
	if (code != hipSuccess) {
        fprintf(stderr, "Cuda error: %s \n in file : %s line number : %d", hipGetErrorString(code), file, line );
        exit(-1);
   }
}


/* Check whether a previous memory allocation was successful. If RAM is full usually the returned value is a NULL pointer.
For example if you allocate memory by doing 
int *mem = malloc(sizeof(int)*SIZE)
check whether it was successful by calling
checkAllocRAM(mem) afterwards */

void checkAllocRAM(void *ptr){
	if (ptr==NULL){
		fprintf(stderr, "Memory Full.\nYour array is too large. Please try a smaller array.\n");
		exit(EXIT_FAILURE);
	}
}

/* This checks whether a file has been opened corrected. If a file opening failed the returned value is a NULL pointer
FOr example if you open a file using
FILE *file=fopen("file.txt","r");
check by calling isFileValid(file); */

void isFileValid(FILE *fp){
	if (fp==NULL){
		perror("A file access error occurred\n");
		exit(EXIT_FAILURE);
	}
}