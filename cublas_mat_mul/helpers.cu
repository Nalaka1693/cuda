/* C file that implements the helper functions specified in helpers.cu */

#include <stdio.h>
#include "helpers.cuh"
#include <hipblas.h>	
#include <hiprand.h>
	
/* check whether the last CUDA function or CUDA kernel launch is erroneous and if yes an error message will be printed
and then the program will be aborted*/

void gpuAssert(const char *file, int line){

	hipError_t code = hipGetLastError();
	if (code != hipSuccess) {
        fprintf(stderr, "Cuda error: %s \n in file : %s line number : %d", hipGetErrorString(code), file, line );
        exit(-1);
   }
}

/*check whether a returned error code by a cublas api function is a erroneous and if yes print the error message*/
void checkCublas(int code,const char *file, int line){
	if(code!=HIPBLAS_STATUS_SUCCESS){
		fprintf(stderr, "Cublas error: %s \n in file : %s line number : %d", cublasGerErrorString(code), file, line );
		exit(-1);
	}

}

/*check whether a returned error code by a hiprand api function is a erroneous and if yes print the error message*/
void checkCurand(int code,const char *file, int line){
	if(code!=HIPBLAS_STATUS_SUCCESS){
		fprintf(stderr, "Cublas error: %s \n in file : %s line number : %d", curandGerErrorString(code), file, line );
		exit(-1);
	}

}

/*Return the error message based the error code for cublas */
static const char *cublasGerErrorString(int error)
{
    switch (error)
    {
        case HIPBLAS_STATUS_SUCCESS:
            return "HIPBLAS_STATUS_SUCCESS";

        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "HIPBLAS_STATUS_NOT_INITIALIZED";

        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "HIPBLAS_STATUS_ALLOC_FAILED";

        case HIPBLAS_STATUS_INVALID_VALUE:
            return "HIPBLAS_STATUS_INVALID_VALUE";

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "HIPBLAS_STATUS_ARCH_MISMATCH";

        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "HIPBLAS_STATUS_MAPPING_ERROR";

        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "HIPBLAS_STATUS_EXECUTION_FAILED";

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "HIPBLAS_STATUS_INTERNAL_ERROR";

        case HIPBLAS_STATUS_NOT_SUPPORTED:
            return "HIPBLAS_STATUS_NOT_SUPPORTED";

        case HIPBLAS_STATUS_UNKNOWN:
            return "HIPBLAS_STATUS_UNKNOWN";
    }

    return "<unknown>";
}

/*Return the error message based the error code for hiprand */
static const char *curandGerErrorString(int error)
{
    switch (error)
    {
        case HIPRAND_STATUS_SUCCESS:
            return "HIPRAND_STATUS_SUCCESS";

        case HIPRAND_STATUS_VERSION_MISMATCH:
            return "HIPRAND_STATUS_VERSION_MISMATCH";

        case HIPRAND_STATUS_NOT_INITIALIZED:
            return "HIPRAND_STATUS_NOT_INITIALIZED";

        case HIPRAND_STATUS_ALLOCATION_FAILED:
            return "HIPRAND_STATUS_ALLOCATION_FAILED";

        case HIPRAND_STATUS_TYPE_ERROR:
            return "HIPRAND_STATUS_TYPE_ERROR";

        case HIPRAND_STATUS_OUT_OF_RANGE:
            return "HIPRAND_STATUS_OUT_OF_RANGE";

        case HIPRAND_STATUS_LENGTH_NOT_MULTIPLE:
            return "HIPRAND_STATUS_LENGTH_NOT_MULTIPLE";

        case HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED:
            return "HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED";

        case HIPRAND_STATUS_LAUNCH_FAILURE:
            return "HIPRAND_STATUS_LAUNCH_FAILURE";

        case HIPRAND_STATUS_PREEXISTING_FAILURE:
            return "HIPRAND_STATUS_PREEXISTING_FAILURE";

        case HIPRAND_STATUS_INITIALIZATION_FAILED:
            return "HIPRAND_STATUS_INITIALIZATION_FAILED";

        case HIPRAND_STATUS_ARCH_MISMATCH:
            return "HIPRAND_STATUS_ARCH_MISMATCH";

        case HIPRAND_STATUS_INTERNAL_ERROR:
            return "HIPRAND_STATUS_INTERNAL_ERROR";
    }

    return "<unknown>";
}

/* Check whether a previous memory allocation was successful. If RAM is full usually the returned value is a NULL pointer.
For example if you allocate memory by doing 
int *mem = malloc(sizeof(int)*SIZE)
check whether it was successful by calling
checkAllocRAM(mem) afterwards */

void checkAllocRAM(void *ptr){
	if (ptr==NULL){
		fprintf(stderr, "Memory Full.\nYour array is too large. Please try a smaller array.\n");
		exit(EXIT_FAILURE);
	}
}

/* This checks whether a file has been opened corrected. If a file opening failed the returned value is a NULL pointer
FOr example if you open a file using
FILE *file=fopen("file.txt","r");
check by calling isFileValid(file); */

void isFileValid(FILE *fp){
	if (fp==NULL){
		perror("A file access error occurred\n");
		exit(EXIT_FAILURE);
	}
}