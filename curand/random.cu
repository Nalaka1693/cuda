/*Generate random floating point matrix using CUDA using hiprand library */

#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>
#include "helpers.cuh"

#define ROWS 100
#define COLS 100
#define SEED 0
#define FILENAME "output.txt"


int main(){

	float *matrix, *matrix_cuda;

	//memory allocation in ram
	matrix=(float *)malloc(sizeof(float)*COLS*ROWS);
	if(matrix==NULL){
		fprintf(stderr,"RAM full\n");
		exit(1);
	}
	
	//memory allocation in cuda
	hipMalloc((void **)&matrix_cuda,sizeof(float)*COLS*ROWS); checkCudaError();
	
	/*create a random number generator.
	Here HIPRAND_RNG_PSEUDO_DEFAULT is the type of the generator
	There are various other types*/
	hiprandGenerator_t generator;	
	checkCurandError(hiprandCreateGenerator(&generator,HIPRAND_RNG_PSEUDO_DEFAULT));
	//set the seed
	checkCurandError(hiprandSetPseudoRandomGeneratorSeed(generator,SEED));
	
	//Time measurement
	hipEvent_t start,stop;
	float elapsedtime;
	hipEventCreate(&start);
	hipEventRecord(start,0);	
	
	/*generate random numbers in a uniform distribution
	There are various other distributions given by functions
	hiprandGenerateNormal, hiprandGenerateLogNormal, hiprandGeneratePoisson
	If you are generating double values use
	hiprandGenerateUniformDouble, hiprandGenerateNormalDouble etc*/
	checkCurandError(hiprandGenerateUniform(generator,matrix_cuda,COLS*ROWS));
	
	//time end
	hipEventCreate(&stop);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedtime,start,stop);
	printf("Time spent for random number generation is : %.10f s\n",elapsedtime/(float)1000);	
	
	//memory copy 
	hipMemcpy(matrix,matrix_cuda,sizeof(float)*COLS*ROWS,hipMemcpyDeviceToHost); checkCudaError();
	
	//writing to file
	FILE *fp=fopen(FILENAME,"w");
	if(fp==NULL){
		fprintf(stderr,"Cannot open file for writing\n");
		exit(1);
	}
	
	int i,j;
	for (i=0;i<ROWS;i++){
		for (j=0;j<COLS;j++){
			fprintf(fp,"%f ",matrix[i*COLS+j]*10);
		}
		fprintf(fp,"\n");
	}	
	
	//free
	hipFree(matrix_cuda); checkCudaError();
	checkCurandError(hiprandDestroyGenerator(generator));
	free(matrix);

	return 0;
}